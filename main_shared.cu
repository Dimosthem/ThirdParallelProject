//Input-------------------------------------------------------------------------------------------------
#define WindowDimension 3						// this is the dimension of the window. 
#define PatchSigma 0.1                          // this is h squared , mentioned in the report
#define Sigma 0.05   							// this is the sigma squared , mentioned in the report
#define FILENAME "images/rasp_noise.csv"		// path to the csv of the image you want to use as input
//End of input------------------------------------------------------------------------------------------




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define sharedSize 24

void printFloatArray(float* arr, int x, int y){
	int i;
	int j;

	for(i=0; i<x; i++){
		for(j=0; j<y; j++){
			printf("%f ", arr[i*y + j]);
		}
		printf("\n");
	}
}
//fast exp , sacrifising some accuracy
__device__ float expf_fast(float a) {
  union { float f; int x; } u;
  u.x = (int) (12102203 * a + 1064866805);
  return u.f;
}



//Calculate the mean of each pixel , using the moving-window technique
__global__ void means( float* pixels , int *windowSize, int* iconLine, float *means)
{
	

	int xIndex = threadIdx.x+ blockIdx.x* blockDim.x;
	int* index = &xIndex;
	int i, j;
	int lim = (*windowSize)/2;
	int dimension = *windowSize;

	float result[WindowDimension*WindowDimension];
	
	for(i=-lim; i<=lim; i++){
		for(j=-lim; j<=lim; j++){
			int location = (*index) + i +j*(*iconLine);

			bool outLeftorRight = (location/(*iconLine) != ((*index)/(*iconLine) + j) ) || location<0;
			bool outUpOrDown = location/(*iconLine) <0 || location/(*iconLine) >= (*iconLine);
			bool ifResult = !(outLeftorRight || outUpOrDown);
			int resultIndex = (j+lim)*(*windowSize) + (i+lim);

			//if the window is completely inside the image
			if(ifResult){ 
				result[resultIndex] = pixels[location];
			}
			
			//if part of the window is outside of the image
			else{
				

				location = (*index) - i -j*(*iconLine);
				
				outLeftorRight =location/(*iconLine) != ((*index)/(*iconLine) - j) || location<0;
				outUpOrDown = location/(*iconLine) <0 || location/(*iconLine) >= (*iconLine);

				if(!outLeftorRight && !outUpOrDown){
					result[resultIndex] = pixels[location];
					continue;
				}

				location = (*index) -i +j*(*iconLine);
				outLeftorRight =location/(*iconLine) != ((*index)/(*iconLine) + j) || location<0;
				outUpOrDown = location/(*iconLine) <0 || location/(*iconLine) >= (*iconLine);

				if(!outLeftorRight && !outUpOrDown){
					result[resultIndex] = pixels[location];
					continue;
				}

				location = (*index) +i -j*(*iconLine);
				result[resultIndex] = pixels[location];
			}

		}	
	}
	float mean=0;
	
	float patchSigma = PatchSigma;
	float tmp = 0;
	for(i=0; i<*windowSize*(*windowSize); i++){
		int x = i%dimension - dimension/2;
		int y = i/dimension - dimension/2;

		float fx = (float)x;
		float fy = (float)y;
		float arithmitis = fx*fx + fy*fy;
		float paronomastis = 2*M_PI*patchSigma;
		
		mean = mean + result[i]*expf_fast(-arithmitis/paronomastis)*0.5;
		tmp +=expf_fast(-arithmitis/paronomastis)*0.5;
	

	}
	
	mean = mean/tmp;
	means[xIndex] = mean; 

}

//denoise the image using the formulas mentioned in the report
__global__ void denoise_shared(float* pixels, float* sigma, int* imageDimension,int* windowDimension,float* means, float* result){
	int xIndex = threadIdx.x+ blockIdx.x* blockDim.x;
	int thread= threadIdx.x;
	int windowSize;
	int imageSize;
	imageSize = *imageDimension*(*imageDimension);
	windowSize = *windowDimension*(*windowDimension); 
	
	__shared__ float shared_pixels[sharedSize];
	__shared__ float shared_means[sharedSize];
	float mean1 = means[xIndex];
	int i=0;
	float sumW = 0 ;
	float sumP = 0;

	for(i=0; i<imageSize; i= i + sharedSize){
		if(i%sharedSize==0){
			if(threadIdx.x<sharedSize)
				shared_pixels[threadIdx.x] = pixels[threadIdx.x+i];
				

		}
		__syncthreads();
		if(i%24==0){
			if(threadIdx.x<sharedSize)
				shared_means[threadIdx.x] = means[threadIdx.x+i];
				

		}
		__syncthreads();
		
		int j =0; 
		for(j=0; j<sharedSize; j++){
			float mean2 = shared_means[j];
			float tmp = (mean1 - mean2)*(mean1-mean2)*(-1);
			float weight = exp(tmp/(*sigma));

			sumW = sumW + weight;
			sumP = sumP + weight*shared_pixels[j];
		}
	}
	sumP = sumP/sumW;
	result[xIndex] = sumP;
	
}

//read the csv and put it in a float array
float* readCSVfile(char* filename, int* dimension){
	FILE *file;
	file = fopen(filename, "r");
	int local_dimension = 256; 
	float* result = (float*)malloc(local_dimension*local_dimension*sizeof(float));  // dont forget to free the memory afterwards
	if(file == NULL){
		printf("The file could not be opened/ does not exits");
		exit(1);
	}

	char* line =NULL;
	size_t length = 0 ;
	ssize_t read;
	const char delimeters[] = ", ";

	char* number ;
	int outer_counter = 0;
	while ((read = getline(&line, &length, file) != -1)){
		
		number = strtok(line, delimeters);
		float fnumber = atof(number);
		int counter = 0;

		while( number !=NULL){
			result[counter + outer_counter*local_dimension] = fnumber;
			counter++;
		

			number = strtok(NULL, delimeters);
			if(number != NULL)	
				fnumber = atof(number);
		}

		if(counter != local_dimension){
			local_dimension = counter;
			
			result =  (float*)realloc(result, counter*counter*sizeof(float));
			
		}


		outer_counter++;
		

	}

	fclose(file);
	if(line)
		free(line);

	*dimension = local_dimension;
	return result;
}



//save a float array to csv
void floatToCSV(char* filename, int dimension, float* arr){
	int i;
	FILE* fileWriter;
	fileWriter = fopen(filename, "a");
	if(fileWriter == NULL){
		printf("something went wrong when saving");
		exit(1);
	}
	for(i=0; i<dimension*dimension ; i++){
	
		if(i!=0 && i%dimension==0)
			fputs("\n", fileWriter);
		if(arr[i]!=arr[i])
			arr[i]=0;
		
		char stringFl[10];
		sprintf(stringFl, "%f", arr[i]);
		fputs( stringFl, fileWriter );
		fputs( ",", fileWriter );
	}
	fclose(fileWriter);
}
int main(void){

	
	float sigma = Sigma;
	int windowDimension = WindowDimension;
	int dimension ; 
	float *image ;
	image = readCSVfile(FILENAME, &dimension);

	
	
	//device pointers 
	float *d_image;
	float *d_sigma;
	int* d_windowDimension;
	int* d_imageDimension;
	float* d_weights;
	float* d_finalImage;
	float* d_means;

	hipMalloc((void**)&d_means, dimension*dimension*sizeof(float));
	hipMalloc((void**)&d_finalImage, dimension*dimension*sizeof(float));

	hipMalloc((void**)&d_image, dimension*dimension*sizeof(float));
	hipMemcpy(d_image, image, dimension*dimension*sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&d_sigma, sizeof(float));
	hipMemcpy(d_sigma	, &sigma, sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_windowDimension, sizeof(int));
	hipMemcpy(d_windowDimension, &windowDimension, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_imageDimension, sizeof(int));
	hipMemcpy(d_imageDimension, &dimension, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_weights, dimension*dimension*sizeof(float));


	

	float* finalPixel = (float*)malloc(sizeof(float)*dimension*dimension); //final result 
	clock_t start, end;
    double cpu_time_used;

    start = clock();
    //the whole algorithm gets executed here
    means<<<dimension, dimension>>>(d_image, d_windowDimension, d_imageDimension, d_means);
    denoise_shared<<<dimension , dimension>>>(d_image, d_sigma, d_imageDimension, d_windowDimension,d_means, d_finalImage);
    //-------------------------------------
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
  	if(error != hipSuccess)
  	{
    // print the CUDA error message and exit
    	printf("CUDA error: %s\n", hipGetErrorString(error));
   		exit(-1);
  	}
    hipMemcpy(finalPixel, d_finalImage, sizeof(float)*dimension*dimension, hipMemcpyDeviceToHost);
	

	end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("program took %f seconds to execute \n", cpu_time_used);

    floatToCSV("imageAfter.csv", dimension, finalPixel);
    
	hipFree(d_weights);
	hipFree(d_image);
	hipFree(d_sigma);
	hipFree(d_windowDimension);
	hipFree(d_imageDimension); 






	return 0; 

}