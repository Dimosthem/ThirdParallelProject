//Input-------------------------------------------------------------------------------------------------
#define WindowDimension 3						// this is the dimension of the window. 
#define PatchSigma 0.01                         // this is h squared , mentioned in the report
#define Sigma 0.05   							// this is the sigma squared , mentioned in the report
#define FILENAME "images/rasp_noise.csv"		// path to the csv of the image you want to use as input
//End of input------------------------------------------------------------------------------------------






#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

//fast exp , sacrifising some accuracy
float expf_fastCPU(float a) {
  union { float f; int x; } u;
  u.x = (int) (12102203 * a + 1064866805);
  return u.f;
}

//Calculate the mean of each pixel , using the moving-window technique
void meansCPU(float* pixels , int *windowSize, int* iconLine, float *means){
	int xIndex = 0;
	for(xIndex = 0; xIndex<*iconLine*(*iconLine); xIndex++){


		int* index = &xIndex;
		int i, j;
		int lim = (*windowSize)/2;
		int dimension = *windowSize;

		float result[WindowDimension*WindowDimension];
		
		for(i=-lim; i<=lim; i++){
			for(j=-lim; j<=lim; j++){
				int location = (*index) + i +j*(*iconLine);

				bool outLeftorRight = (location/(*iconLine) != ((*index)/(*iconLine) + j) ) || location<0;
				bool outUpOrDown = location/(*iconLine) <0 || location/(*iconLine) >= (*iconLine);
				bool ifResult = !(outLeftorRight || outUpOrDown);
				int resultIndex = (j+lim)*(*windowSize) + (i+lim);

				//if the window is completely inside the image
				if(ifResult){
					result[resultIndex] = pixels[location];
				}
				
		 		//if part of the window is outside of the image
				else{
		

					location = (*index) - i -j*(*iconLine);
					
					outLeftorRight =location/(*iconLine) != ((*index)/(*iconLine) - j) || location<0;
					outUpOrDown = location/(*iconLine) <0 || location/(*iconLine) >= (*iconLine);

					if(!outLeftorRight && !outUpOrDown){
						result[resultIndex] = pixels[location];
						continue;
					}

					location = (*index) -i +j*(*iconLine);
					outLeftorRight =location/(*iconLine) != ((*index)/(*iconLine) + j) || location<0;
					outUpOrDown = location/(*iconLine) <0 || location/(*iconLine) >= (*iconLine);

					if(!outLeftorRight && !outUpOrDown){
						result[resultIndex] = pixels[location];
						continue;
					}

					location = (*index) +i -j*(*iconLine);
					result[resultIndex] = pixels[location];
				}

			}	
		}
		float mean=0;
		
		float patchSigma = PatchSigma;
		float tmp = 0;
		for(i=0; i<*windowSize*(*windowSize); i++){
			int x = i%dimension - dimension/2;
			int y = i/dimension - dimension/2;

			float fx = (float)x;
			float fy = (float)y;
			float arithmitis = fx*fx + fy*fy;
			float paronomastis = 2*M_PI*patchSigma;
			
			mean = mean + result[i]*expf_fastCPU(-arithmitis/paronomastis)*0.5;
			tmp = tmp+expf_fastCPU(-arithmitis/paronomastis)*0.5;
		
		}
		
		mean = mean/tmp;
		means[xIndex] = mean; 
	}
}
//denoise the image using the formulas mentioned in the report
void denoiseCPU(float* pixels, float* sigma, int* imageDimension,int* windowDimension,float* means, float* result){
	int xIndex =0;
	for(xIndex=0; xIndex<*imageDimension*(*imageDimension); xIndex++){
		int windowSize;
		int imageSize;
		imageSize = *imageDimension*(*imageDimension);
		windowSize = *windowDimension*(*windowDimension); 
		float mean1 = means[xIndex];
		int i=0;
		float sumW = 0 ;
		float sumP = 0;
		for(i=0; i<imageSize; i++){
			float mean2 = means[i];
		
			float tmp = (mean1 - mean2)*(mean1-mean2)*(-1);
			float weight = exp(tmp/(*sigma));

			sumW = sumW + weight;
			sumP = sumP + weight*pixels[i];

		}
	sumP = sumP/sumW;
	result[xIndex] = sumP;
	}
}



//read the csv and put it in a float array
float* readCSVfile(char* filename, int* dimension){
	FILE *file;
	file = fopen(filename, "r");
	int local_dimension = 256; 
	float* result = (float*)malloc(local_dimension*local_dimension*sizeof(float));  // dont forget to free the memory afterwards
	if(file == NULL){
		printf("The file could not be opened/ does not exits");
		exit(1);
	}

	char* line =NULL;
	size_t length = 0 ;
	ssize_t read;
	const char delimeters[] = ", ";

	char* number ;
	int outer_counter = 0;
	while ((read = getline(&line, &length, file) != -1)){
		
		number = strtok(line, delimeters);
		float fnumber = atof(number);
		int counter = 0;

		while( number !=NULL){
			result[counter + outer_counter*local_dimension] = fnumber;
			counter++;
			

			number = strtok(NULL, delimeters);
			if(number != NULL)	
				fnumber = atof(number);
		}

		if(counter != local_dimension){
			local_dimension = counter;
			
			result =  (float*)realloc(result, counter*counter*sizeof(float));
			
		}


		outer_counter++;
		

	}

	fclose(file);
	if(line)
		free(line);

	*dimension = local_dimension;
	return result;
}



//save a float array to csv
void floatToCSV(char* filename, int dimension, float* arr){
	int i;
	FILE* fileWriter;
	fileWriter = fopen(filename, "a");
	if(fileWriter == NULL){
		printf("something went wrong when saving");
		exit(1);
	}
	for(i=0; i<dimension*dimension ; i++){
	
		if(i!=0 && i%dimension==0)
			fputs("\n", fileWriter);
		if(arr[i]!=arr[i])
			arr[i]=0;
	

	 
		char stringFl[10];
		sprintf(stringFl, "%f", arr[i]);
		fputs( stringFl, fileWriter );
		fputs( ",", fileWriter );
	}
	fclose(fileWriter);
}
int main(void){

	
	float sigma = Sigma;
	int windowDimension = WindowDimension;
	int dimension ; 
	float *image ;
	image = readCSVfile(FILENAME, &dimension);

	float* means = (float*)malloc(dimension*dimension*sizeof(float));

	
	
	
	


	

	float* finalPixel = (float*)malloc(sizeof(float)*dimension*dimension);
	clock_t start, end;
    double cpu_time_used;
    start = clock();

    //the whole algorithm gets executed here
    meansCPU(image, &windowDimension, &dimension, means);

    denoiseCPU(image, &sigma, &dimension, &windowDimension, means, finalPixel);
	//-------------------------------------

	end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("program took %f seconds to execute \n", cpu_time_used);

    floatToCSV("imageAfter.csv", dimension, finalPixel);
    
	








	

	return 0; 

}